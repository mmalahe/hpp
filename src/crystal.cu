#include "hip/hip_runtime.h"
#include <hpp/tensorCUDA.h>
#include <hpp/crystalCUDA.h>

#include <hpp/spectralUtilsCUDA.h>

#include <hpp/hdfUtilsCpp.h>

namespace hpp
{
#ifdef HPP_USE_CUDA

template <typename T, unsigned int N>
CrystalPropertiesCUDA<T,N>::CrystalPropertiesCUDA(const CrystalProperties<T>& in) {
    // Check compatability
    if (in.n_alpha != N) {
        throw CrystalError("Mismatch in number of slip systems.");
    }
    
    // Populate scalars
    n_alpha = in.n_alpha;
    mu = in.mu;
    kappa = in.kappa;
    m = in.m;
    gammadot_0 = in.gammadot_0;
    h_0 = in.h_0;
    s_s = in.s_s;
    a = in.a;
    q = in.q;
    volume = in.volume;
    
    // Populate vectors and tensors
    for (unsigned int i=0; i<n_alpha; i++) {
        m_0[i] = VecCUDA<T,3>(in.m_0[i]);
        n_0[i] = VecCUDA<T,3>(in.n_0[i]);
        S_0[i] = Tensor2CUDA<T,3,3>(in.S_0[i]);
    }
    L = Tensor4CUDA<T,3,3,3,3>(in.L);
    Q = Tensor2CUDA<T,N,N>(in.Q);
}

template <typename T>
SpectralCrystalListCUDA<T>::SpectralCrystalListCUDA(unsigned int nCrystals, const SpectralCrystalCUDA<T> *crystals) {
    // Assign the values
    std::vector<T> anglesAH(nCrystals);
    std::vector<T> anglesBH(nCrystals);
    std::vector<T> anglesCH(nCrystals);
    std::vector<T> sH(nCrystals);
    for (unsigned int iCrystal=0; iCrystal<nCrystals; iCrystal++) {
        anglesAH[iCrystal] = crystals[iCrystal].angles.alpha;
        anglesBH[iCrystal] = crystals[iCrystal].angles.beta;
        anglesCH[iCrystal] = crystals[iCrystal].angles.gamma;
        sH[iCrystal] = crystals[iCrystal].s;
    }
    
    // Make the device copies
    sharedPtrs.push_back(makeDeviceCopyVecSharedPtr(anglesAH));
    anglesA = sharedPtrs.back().get();
    
    sharedPtrs.push_back(makeDeviceCopyVecSharedPtr(anglesBH));
    anglesB = sharedPtrs.back().get();
    
    sharedPtrs.push_back(makeDeviceCopyVecSharedPtr(anglesCH));
    anglesC = sharedPtrs.back().get();
    
    sharedPtrs.push_back(makeDeviceCopyVecSharedPtr(sH));
    s = sharedPtrs.back().get();
}

template <typename T>
SpectralCrystalListCUDA<T>::SpectralCrystalListCUDA(const std::vector<SpectralCrystalCUDA<T>>& crystals) {
    *this = SpectralCrystalListCUDA(crystals.size(), crystals.data());
}

template <typename T, unsigned int N>
void SpectralPolycrystalCUDA<T,N>::doCrystalSetup(const std::vector<SpectralCrystalCUDA<T>>& crystals, const CrystalPropertiesCUDA<T, N>& crystalProps) {
    // Establish the CUDA context
    CUDA_CHK(hipFree(0));
    
    // Direct device copies
    nCrystals = crystals.size();
    if (nCrystals % 2 != 0) {
        std::cerr << "Warning: paired crystal implementation does not account for odd total number of crystals" << std::endl;
        ///@todo account for this
    }
    nCrystalPairs = nCrystals/2;
    crystalsD = makeDeviceCopyVecSharedPtr(crystals);
    
    // Crystal properties    
    crystalPropsD = makeDeviceCopySharedPtr(crystalProps);
    
    // Initialise global cauchy stress
    TCauchyGlobalD = makeDeviceCopySharedPtr(this->TCauchyGlobalH);
}

template <typename T, unsigned int N>
void SpectralPolycrystalCUDA<T,N>::doGPUSetup() {
    // Find how many GPUs are avaiable
    int nDevices;
    CUDA_CHK(hipGetDeviceCount(&nDevices));
    if (nDevices < 1) {
        throw std::runtime_error("No GPU to use.");
    }
    
    // Select the first GPU
    deviceID = 0;   
    CUDA_CHK(hipGetDeviceProperties(&devProp, deviceID));
    std::cout << "Using " << devProp.name << std::endl;
    
    // Get parallel layout for step kernel
    if (useUnifiedDB) {
        stepKernelCfg = getKernelConfigMaxOccupancy(devProp, (void*)SPECTRAL_POLYCRYSTAL_STEP_UNIFIED<T,N,9>, nCrystalPairs);
    }
    else {
        stepKernelCfg = getKernelConfigMaxOccupancy(devProp, (void*)SPECTRAL_POLYCRYSTAL_STEP<T,N>, nCrystals);
    }
    unsigned int nBlocks = stepKernelCfg.dG.x;
    std::cout << "Step kernel:" << std::endl;
    std::cout << stepKernelCfg;
    
    // Get parallel layout for reduce kernel
    reduceKernelLevel0Cfg = getKernelConfigMaxOccupancy(devProp, (void*)BLOCK_REDUCE_KEPLER_TENSOR2<T,3,3>, nBlocks);
    std::cout << "Reduce kernel level 0:" << std::endl;
    std::cout << reduceKernelLevel0Cfg;
    
    // Check if we need a second level of reduction
    if (reduceKernelLevel0Cfg.dG.x > 1) {        
        reduceKernelLevel1Cfg = getKernelConfigMaxOccupancy(devProp, (void*)BLOCK_REDUCE_KEPLER_TENSOR2<T,3,3>, reduceKernelLevel0Cfg.dG.x);
        TCauchyLevel0Sums = allocDeviceMemorySharedPtr<Tensor2CUDA<T,3,3>>(reduceKernelLevel0Cfg.dG.x);
        std::cout << "Reduce kernel level 1:" << std::endl;
        std::cout << reduceKernelLevel1Cfg;
    }
    else {
        reduceKernelLevel1Cfg.dG.x = 0;
        reduceKernelLevel1Cfg.dG.y = 0;
        reduceKernelLevel1Cfg.dG.z = 0;
        reduceKernelLevel1Cfg.dB.x = 0;
        reduceKernelLevel1Cfg.dB.y = 0;
        reduceKernelLevel1Cfg.dB.z = 0;
    }

    // Working memory
    TCauchyPerBlockSums = allocDeviceMemorySharedPtr<Tensor2CUDA<T,3,3>>(nBlocks);
    
    // Memory report
    double usedGiB = getUsedMemoryGiB();
    std::cout << "Used Memory (GiB) = " << usedGiB << std::endl;    
}

// Main constructors
template <typename T, unsigned int N>
SpectralPolycrystalCUDA<T,N>::SpectralPolycrystalCUDA(const std::vector<SpectralCrystalCUDA<T>>& crystals, const CrystalPropertiesCUDA<T, N>& crystalProps, const SpectralDatabase<T>& dbIn){    
    // Not using unified database
    useUnifiedDB = false;
    
    // Set up crystals
    this->doCrystalSetup(crystals, crystalProps);
    
    // Set up GPU parameters based on crystals and available hardware
    this->doGPUSetup();
    
    // Set up database
    std::vector<SpectralDatasetID> dsetIDs = defaultCrystalSpectralDatasetIDs();
    dbH = SpectralDatabaseCUDA<T,4>(dbIn, dsetIDs);    
    dbD = makeDeviceCopySharedPtr(this->dbH);
}

template <typename T, unsigned int N>
SpectralPolycrystalCUDA<T,N>::SpectralPolycrystalCUDA(const std::vector<SpectralCrystalCUDA<T>>& crystals, const CrystalPropertiesCUDA<T, N>& crystalProps, const SpectralDatabaseUnified<T>& dbIn){
    // Using unified database
    useUnifiedDB = true;
    
    // Set up crystals
    this->doCrystalSetup(crystals, crystalProps);
    
    // Set up GPU parameters
    this->doGPUSetup();
    
    // Set up database
    std::vector<SpectralDatasetID> dsetIDs = defaultCrystalSpectralDatasetIDs();    
    dbUnifiedH = SpectralDatabaseUnifiedCUDA<T,4,9>(dbIn, dsetIDs);    
    dbUnifiedD = makeDeviceCopySharedPtr(this->dbUnifiedH);
    
    // Get memory usage
    maxMemUsedGB = getUsedMemoryGB();
}

template <typename T>
__device__ Tensor2CUDA<T,3,3> getSigmaPrime(T sigmaScaling, T *dbVars) {
    Tensor2CUDA<T,3,3> sigmaPrime;
    
    // Only the upper triangular terms of sigmaPrime
    sigmaPrime(0,0) = sigmaScaling*dbVars[SIGMA00];
    sigmaPrime(1,1) = sigmaScaling*dbVars[SIGMA11];
    sigmaPrime(2,2) = -sigmaPrime(0,0) -sigmaPrime(1,1);//deviatoric component
    sigmaPrime(1,2) = sigmaScaling*dbVars[SIGMA12];
    sigmaPrime(0,2) = sigmaScaling*dbVars[SIGMA02];
    sigmaPrime(0,1) = sigmaScaling*dbVars[SIGMA01];    
    
    // Symmetric terms
    sigmaPrime(2,1) = sigmaScaling*sigmaPrime(1,2);
    sigmaPrime(2,0) = sigmaScaling*sigmaPrime(0,2);
    sigmaPrime(1,0) = sigmaScaling*sigmaPrime(0,1);
    
    // Return
    return sigmaPrime;
}

template <typename T>
__device__ Tensor2AsymmCUDA<T,3> getWp(T WpScaling, T*dbVars) {
    Tensor2AsymmCUDA<T,3> Wp;
    
    // Only the terms (0,1), (1,2) and (0,2)
    // Anti-symmetric terms are handled internally by Tensor2AsymmCUDA
    Wp.setVal(0,1,WpScaling*dbVars[WP01]);
    Wp.setVal(1,2,WpScaling*dbVars[WP12]);
    Wp.setVal(0,2,WpScaling*dbVars[WP02]);
    
    // Return
    return Wp;
}

// Step kernel
template<typename T, unsigned int N>
__global__ void SPECTRAL_POLYCRYSTAL_STEP(unsigned int nCrystals, SpectralCrystalCUDA<T>* crystals, CrystalPropertiesCUDA<T,N>* props, 
Tensor2CUDA<T,3,3> RStretchingTensor, Tensor2AsymmCUDA<T,3> WNext, T theta, T strainIncrement, T dt, SpectralDatabaseCUDA<T,4>* db, Tensor2CUDA<T,3,3> *TCauchyPerBlockSums) 
{
    // Get crystal index
    unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;
    
    // Shared memory for IDFT
    const unsigned int nSharedSpectral = 1024;
    __shared__ SpectralCoordCUDA<4> sharedCoords[nSharedSpectral];
    __shared__ SpectralCoeffCUDA<T> sharedCoeffs[nSharedSpectral];
    
    // If out of bounds, go through motions of calculation, but don't update at end
    /// @fixme: extend this approach to the Cauchy stress sum at the end
    bool doUpdateCrystalState = true;
    if (idx > nCrystals-1) {
        idx = nCrystals-1;
        doUpdateCrystalState = false;
    }
    
    // Get copy of our crystal from global memory
    SpectralCrystalCUDA<T> crystal = crystals[idx];
   
    // The rotation that transforms the template crystal to have the same orientation as this one
    // First, the rotation to get to the initial configuration: init.crystalRotation
    // Second, the further rotation caused by the deformation: RStar
    Tensor2CUDA<T,3,3> R = EulerZXZRotationMatrixCUDA(crystal.angles);
    
    // Transform into the stretching tensor frame
    R = RStretchingTensor.trans()*R;
    
    // Euler angles
    EulerAngles<T> angles = getEulerZXZAngles(R);
    
    // There are possible branches in getEulerZXZAngles, so sync threads here
    // to head off divergence.
    __syncthreads();

    // Database coordinate
    T gridPos[4] = {angles.alpha, angles.beta, angles.gamma, theta};
    unsigned int spatialCoord[4];
    T *gridStarts = db->getGridStarts();
    T *gridSteps = db->getGridSteps();
    for (unsigned int i=0; i<4; i++) {
        spatialCoord[i] = (unsigned int) ((gridPos[i] - gridStarts[i])/gridSteps[i]);
    }
    
    // Variables to fetch
    Tensor2CUDA<T,3,3> sigmaPrimeNext;
    Tensor2AsymmCUDA<T,3> WpNext;
    T gammaNext;
    
    // Gamma
    T gammaScaling = strainIncrement;
    gammaNext = gammaScaling*db->getIDFTRealDShared(GAMMA, spatialCoord, nSharedSpectral, sharedCoords, sharedCoeffs);
    
    // Update slip deformation resistance
    crystal.s = slipDeformationResistanceStepSpectralSolver(props, crystal.s, gammaNext, dt);
    
    // Sigma
    T sigmaScaling = (crystal.s*powIntrinsic(fabs(strainIncrement), props->m));
    
    // Only the upper triangular terms of sigmaPrime
    sigmaPrimeNext(0,0) = sigmaScaling*db->getIDFTRealDShared(SIGMA00, spatialCoord, nSharedSpectral, sharedCoords, sharedCoeffs);
    sigmaPrimeNext(1,1) = sigmaScaling*db->getIDFTRealDShared(SIGMA11, spatialCoord, nSharedSpectral, sharedCoords, sharedCoeffs);
    sigmaPrimeNext(2,2) = -sigmaPrimeNext(0,0) -sigmaPrimeNext(1,1);//deviatoric component
    sigmaPrimeNext(1,2) = sigmaScaling*db->getIDFTRealDShared(SIGMA12, spatialCoord, nSharedSpectral, sharedCoords, sharedCoeffs);
    sigmaPrimeNext(0,2) = sigmaScaling*db->getIDFTRealDShared(SIGMA02, spatialCoord, nSharedSpectral, sharedCoords, sharedCoeffs);
    sigmaPrimeNext(0,1) = sigmaScaling*db->getIDFTRealDShared(SIGMA01, spatialCoord, nSharedSpectral, sharedCoords, sharedCoeffs);    
    
    // Symmetric terms
    sigmaPrimeNext(2,1) = sigmaScaling*sigmaPrimeNext(1,2);
    sigmaPrimeNext(2,0) = sigmaScaling*sigmaPrimeNext(0,2);
    sigmaPrimeNext(1,0) = sigmaScaling*sigmaPrimeNext(0,1);    
 
    // Wp
    T WpScaling = strainIncrement;
    
    // Only the terms (0,1), (1,2) and (0,2)
    WpNext.setVal(0,1,WpScaling*db->getIDFTRealDShared(WP01, spatialCoord, nSharedSpectral, sharedCoords, sharedCoeffs));
    WpNext.setVal(1,2,WpScaling*db->getIDFTRealDShared(WP12, spatialCoord, nSharedSpectral, sharedCoords, sharedCoeffs));
    WpNext.setVal(0,2,WpScaling*db->getIDFTRealDShared(WP02, spatialCoord, nSharedSpectral, sharedCoords, sharedCoeffs));
    
    // Transform into lab frame
    Tensor2CUDA<T,3,3> TCauchy = transformOutOfFrame(sigmaPrimeNext, RStretchingTensor);
    Tensor2CUDA<T,3,3> WpNextLab = transformOutOfFrame(WpNext, RStretchingTensor);
    
    // Update lattice rotation tensor
    Tensor2CUDA<T,3,3> WStarNext = WNext - WpNextLab;
    Tensor2CUDA<T,3,3> RStar = EulerZXZRotationMatrixCUDA(crystal.angles);
    Tensor2CUDA<T,3,3> RStarNext = RStar + WStarNext*RStar*dt;
    
    // Update crystal rotations
    crystal.angles = getEulerZXZAngles(RStarNext);
    
    // There are possible branches in getEulerZXZAngles, so sync threads here
    // to head off divergence.
    __syncthreads();
    
    // Add up the Cauchy stresses for this block    
    Tensor2CUDA<T,3,3> TCauchyBlockSum = TCauchy;
    __syncthreads();
    TCauchyBlockSum = blockReduceSumTensor2(TCauchyBlockSum);
    if (threadIdx.x==0) {
        TCauchyPerBlockSums[blockIdx.x]=TCauchyBlockSum;
    }

    // Restore crystal to global memory
    if (doUpdateCrystalState) {
        crystals[idx] = crystal;
    }
}

// Step kernel
template<typename T, unsigned int N, unsigned int P>
__global__ void SPECTRAL_POLYCRYSTAL_STEP_UNIFIED(unsigned int nCrystalPairs, SpectralCrystalCUDA<T>* crystals, CrystalPropertiesCUDA<T,N>* props, 
Tensor2CUDA<T,3,3> RStretchingTensor, Tensor2AsymmCUDA<T,3> WNext, T theta, T strainIncrement, T dt, SpectralDatabaseUnifiedCUDA<T,4,P>* db, Tensor2CUDA<T,3,3> *TCauchyPerBlockSums) 
{
    // Get crystal index
    unsigned int pairIdx = blockDim.x*blockIdx.x + threadIdx.x;
    
    // Shared memory for IDFT
    const unsigned int nSharedSpectral = (8/sizeof(T))*128;
    __shared__ SpectralDataUnifiedCUDA<T,4,P> sharedData[nSharedSpectral];
    
    // If out of bounds, go through motions of calculation, but don't update at end
    /// @fixme: extend this approach to the Cauchy stress sum at the end
    bool doUpdateCrystalStates = true;
    if (pairIdx > nCrystalPairs-1) {
        pairIdx = nCrystalPairs-1;
        doUpdateCrystalStates = false;
    }
    
    // Indices of each crystal in the pair
    unsigned int idx0 = 2*pairIdx;
    unsigned int idx1 = idx0+1;
    
    // Get copies of the crystals from global memory
    SpectralCrystalCUDA<T> crystal0 = crystals[idx0];
    SpectralCrystalCUDA<T> crystal1 = crystals[idx1];
    
    // Get the correct database coordinate based on the crystal orientations
    unsigned int dbCoord0[4];
    getSpectralCrystalDatabaseCoordinate(crystal0, db, RStretchingTensor, theta, &(dbCoord0[0]));
    unsigned int dbCoord1[4];
    getSpectralCrystalDatabaseCoordinate(crystal1, db, RStretchingTensor, theta, &(dbCoord1[0]));
    
    // There are possible branches in the coordinate fetch, so sync threads here
    // to head off divergence
    __syncthreads();
    
    // Fetch the variables
    T dbVars0[P];
    T dbVars1[P];
    db->getIDFTRealDSharedPair(dbCoord0, dbVars0, dbCoord1, dbVars1, nSharedSpectral, sharedData);
    
    // Gamma
    T gammaScaling = strainIncrement;
    T gammaNext0 = gammaScaling*dbVars0[GAMMA];
    T gammaNext1 = gammaScaling*dbVars1[GAMMA];
    
    // Update slip deformation resistance
    crystal0.s = slipDeformationResistanceStepSpectralSolver(props, crystal0.s, gammaNext0, dt);
    crystal1.s = slipDeformationResistanceStepSpectralSolver(props, crystal1.s, gammaNext1, dt);
    
    // Sigma
    T sigmaScaling0 = (crystal0.s*powIntrinsic(fabs(strainIncrement), props->m));
    T sigmaScaling1 = (crystal1.s*powIntrinsic(fabs(strainIncrement), props->m));
    Tensor2CUDA<T,3,3> sigmaPrimeNext0 = transformOutOfFrame(getSigmaPrime(sigmaScaling0, dbVars0), RStretchingTensor);
    Tensor2CUDA<T,3,3> sigmaPrimeNext1 = transformOutOfFrame(getSigmaPrime(sigmaScaling1, dbVars1), RStretchingTensor);
 
    // Wp
    T WpScaling = strainIncrement;
    Tensor2CUDA<T,3,3> WpNext0 = transformOutOfFrame(getWp(WpScaling, dbVars0), RStretchingTensor);
    Tensor2CUDA<T,3,3> WpNext1 = transformOutOfFrame(getWp(WpScaling, dbVars1), RStretchingTensor);
    
    // Fetch current rotational component of deformation
    Tensor2CUDA<T,3,3> RStar0 = EulerZXZRotationMatrixCUDA(crystal0.angles);
    Tensor2CUDA<T,3,3> RStar1 = EulerZXZRotationMatrixCUDA(crystal1.angles);
    
    // Update lattice rotation tensor
    Tensor2CUDA<T,3,3> WStarNext0 = WNext - WpNext0;
    Tensor2CUDA<T,3,3> WStarNext1 = WNext - WpNext1;
    Tensor2CUDA<T,3,3> RStarNext0 = RStar0 + WStarNext0*RStar0*dt;
    Tensor2CUDA<T,3,3> RStarNext1 = RStar1 + WStarNext1*RStar1*dt;
    
    // Update crystal rotations
    crystal0.angles = getEulerZXZAngles(RStarNext0);
    crystal1.angles = getEulerZXZAngles(RStarNext1);
    
    // There are possible branches in getEulerZXZAngles, so sync threads here
    // to head off divergence.
    __syncthreads();
    
    // Calculate Cauchy stress. 
    // Transform out of the stretching tensor frame
    Tensor2CUDA<T,3,3> pairTCauchySum;
    pairTCauchySum += sigmaPrimeNext0;
    pairTCauchySum += sigmaPrimeNext1;
    
    // Add up the Cauchy stresses for this block    
    Tensor2CUDA<T,3,3> TCauchyBlockSum = pairTCauchySum;
    __syncthreads();
    TCauchyBlockSum = blockReduceSumTensor2(TCauchyBlockSum);
    if (threadIdx.x==0) {
        TCauchyPerBlockSums[blockIdx.x]=TCauchyBlockSum;
    }

    // Restore crystal to global memory
    if (doUpdateCrystalStates) {
        crystals[idx0] = crystal0;
        crystals[idx1] = crystal1;
    }
}

// Average kernel
/**
 * @brief Get the average Cauchy stress.
 * @detail @todo Write a good parallel reduction.
 * @param F_next
 * @param L_next
 * @param dt
 */
template<typename T>
__global__ void GET_AVERAGE_TCAUCHY(unsigned int nCrystals, const SpectralCrystalCUDA<T>* crystals, Tensor2CUDA<T,3,3> *TCauchyGlobal) {
    // Get absolute thread index
    unsigned int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx > 1) return;
    
    // Zero the global memory
    for (unsigned int i=0; i<3; i++) {
        for (unsigned int j=0; j<3; j++) {
            (*TCauchyGlobal)(i,j) = (T)0.0;
        }
    }
    
    // Add it up
    for (unsigned int iCrystal=0; iCrystal<nCrystals; iCrystal++) {
        (*TCauchyGlobal) += crystals[iCrystal].TCauchy;
    }
    
    // Average
    *TCauchyGlobal /= (T)nCrystals;    
}

template<typename T, unsigned int N>
__global__ void HISTOGRAM_POLES_EQUAL_AREA(unsigned int nCrystals, const SpectralCrystalCUDA<T>* crystals, VecCUDA<T,3>* planeNormalG, Tensor2CUDA<T,N,N>* histG) {
    // Get absolute thread index
    unsigned int baseIdx = blockDim.x*blockIdx.x + threadIdx.x;
    
    // Read in normal from global memory
    VecCUDA<T,3> planeNormal = *planeNormalG;

    // Maximum R value from northern hemisphere projection
    T maxR = (1.00001)*2*sinIntrinsic(M_PI/4);
    
    // Grid stride loop over crystals
    for (unsigned int idx=baseIdx; idx<nCrystals; idx+=blockDim.x*gridDim.x) {
        // Get orientation of the crystal
        SpectralCrystalCUDA<T> crystal = crystals[idx];
        Tensor2CUDA<T,3,3> ROrientation = EulerZXZRotationMatrixCUDA(crystal.angles);
        
        // Active rotation
        VecCUDA<T,3> pole = ROrientation*planeNormal;
        VecCUDA<T,3> poleSpherical = cartesianToSpherical(pole);
        T theta = poleSpherical(1);
        T phi = poleSpherical(2);
        
        // Equal-area projection
        T R = 2*sinIntrinsic(phi/2);
        T x, y;
        sincosIntrinsic(theta, &y, &x);
        x *= R;
        y *= R;
        
        // Histogram index
        T xMin = -maxR;
        T xMax = maxR;
        T yMin = xMin;
        T yMax = xMax;
        T binwidthX = (xMax-xMin)/N;
        T binwidthY = (yMax-yMin)/N;
        int ix = (int) ((x-xMin)/binwidthX);
        int iy = (int) ((y-yMin)/binwidthY);
        
        // Add points to histogram
        if (ix >=0 && ix < N && iy>=0 && iy < N) {
            atomicAdd(&((*histG)(ix,iy)), 1.0);
        }
        __syncthreads();
    }    
} 

// Step host function
template <typename T, unsigned int N>
void SpectralPolycrystalCUDA<T,N>::step(const hpp::Tensor2<T>& F_next, const hpp::Tensor2<T>& L_next, T dt) 
{        
    // Get stretching tensor decomposition
    StretchingTensorDecomposition<T> stretchingTensorDecomp = getStretchingTensorDecomposition(L_next); 
    T theta = stretchingTensorDecomp.theta;
    T strainRate = stretchingTensorDecomp.DNorm;
    T strainIncrement = strainRate*dt;
    
    // The rotational component of the stretching tensor
    Tensor2CUDA<T,3,3> RStretchingTensor(stretchingTensorDecomp.evecs);
    
    // The overall spin tensor
    Tensor2AsymmCUDA<T,3> WNext = (T)0.5*(L_next-L_next.trans());
    
    // Compute the next step
    dim3 dG = stepKernelCfg.dG;
    dim3 dB = stepKernelCfg.dB;
    
    // Start solve timer
    solveTimer.start();
    
    if (useUnifiedDB) {
        SPECTRAL_POLYCRYSTAL_STEP_UNIFIED<<<dG,dB>>>(nCrystalPairs, crystalsD.get(), crystalPropsD.get(), RStretchingTensor, WNext, theta, strainIncrement, dt, dbUnifiedD.get(), TCauchyPerBlockSums.get());
    }
    else {
        SPECTRAL_POLYCRYSTAL_STEP<<<dG,dB>>>(nCrystals, crystalsD.get(), crystalPropsD.get(), RStretchingTensor, WNext, theta, strainIncrement, dt, dbD.get(), TCauchyPerBlockSums.get());    
    }
    
    // Stop solve timer
    CUDA_CHK(hipDeviceSynchronize());
    solveTimer.stop();
    
    // Sum up the per-block stress
    // TCauchyGlobalD will now contain the global sum of the stress
    unsigned int nBlocks = stepKernelCfg.dG.x;
    
    // Single level reduction
    if (reduceKernelLevel0Cfg.dG.x <= 1) {
        BLOCK_REDUCE_KEPLER_TENSOR2<<<reduceKernelLevel0Cfg.dG, reduceKernelLevel0Cfg.dB>>>(TCauchyPerBlockSums.get(), TCauchyGlobalD.get(), nBlocks);
    }
    // Two level reduction
    else{        
        BLOCK_REDUCE_KEPLER_TENSOR2<<<reduceKernelLevel0Cfg.dG, reduceKernelLevel0Cfg.dB>>>(TCauchyPerBlockSums.get(), TCauchyLevel0Sums.get(), nBlocks);
        BLOCK_REDUCE_KEPLER_TENSOR2<<<reduceKernelLevel1Cfg.dG, reduceKernelLevel1Cfg.dB>>>(TCauchyLevel0Sums.get(), TCauchyGlobalD.get(), reduceKernelLevel0Cfg.dG.x);
    }
    
    // Sync device and host before copying across memory
    CUDA_CHK(hipDeviceSynchronize());
    
    // Move required quantities to host
    TCauchyGlobalH = getHostValue(TCauchyGlobalD)/(T)nCrystals;
}

template <typename T, unsigned int N>
void SpectralPolycrystalCUDA<T,N>::evolve(T tStart, T tEnd, T dt, std::function<hpp::Tensor2<T>(T t)> F_of_t, std::function<hpp::Tensor2<T>(T t)> L_of_t) {
    // Initial data
    tHistory.push_back(tStart);
    TCauchyHistory.push_back(TCauchyGlobalH);
    
    // Stepping
    unsigned int nsteps = (tEnd-tStart)/dt;    
    for (unsigned int i=0; i<nsteps; i++) {
        // Inputs for the next step
        T t = tStart + (i+1)*dt;
        std::cout << "t = " << t << std::endl;
        hpp::Tensor2<T> LNext = L_of_t(t);     
        hpp::Tensor2<T> FNext = F_of_t(t);
        
        // Step
        this->step(FNext, LNext, dt);
        
        // Store quantities
        tHistory.push_back(t);
        TCauchyHistory.push_back(TCauchyGlobalH);        
    }
    
    // Report clock rate
    std::cout << "Clock rate (GHz) = " << getClockRateGHz(deviceID) << std::endl;
}

/**
 * @brief Writes out pole histograms as a Python dictionary
 * @detail Keys are pole specifiers of the form '110' for the pole 110. Values
 * are the pole histograms.
 * @param outfile the output file
 * @param poles the poles to plot
 */
template <typename T, unsigned int N>
void SpectralPolycrystalCUDA<T,N>::writePoleHistogramsHDF5(H5::H5File& outfile, std::string dsetBaseName, const std::vector<VecCUDA<T,3>>& poles) {
    // Histogram configuration
    const unsigned int histDim = 1024;
    CudaKernelConfig histKernelCfg = getKernelConfigMaxOccupancy(devProp, (void*)HISTOGRAM_POLES_EQUAL_AREA<T, histDim>, nCrystals);
    dim3 dG = histKernelCfg.dG;
    dim3 dB = histKernelCfg.dB;
    
    // Begin writing
    for (unsigned int i=0; i<poles.size(); i++) {        
        // Generate histogram
        auto poleH = poles[i];
        std::shared_ptr<VecCUDA<T,3>> poleD = makeDeviceCopySharedPtr(poleH);    
        Tensor2CUDA<T,histDim,histDim>* histHPtr = new Tensor2CUDA<T,histDim,histDim>;
        std::shared_ptr<Tensor2CUDA<T,histDim,histDim>> histD = makeDeviceCopySharedPtrFromPtr(histHPtr);        
        CUDA_CHK(hipDeviceSynchronize());
        HISTOGRAM_POLES_EQUAL_AREA<T, histDim><<<dG,dB>>>(nCrystals, crystalsD.get(), poleD.get(), histD.get());
        CUDA_CHK(hipDeviceSynchronize());
        copyToHost(histD, histHPtr);
        
        // Write histogram
        std::string dsetName = dsetBaseName + "_";        
        for (auto val : poleH) {
            dsetName += std::to_string((int)val);
        }
        std::vector<hsize_t> dataDims = {histDim, histDim};
        auto dset = createHDF5Dataset<T>(outfile, dsetName, dataDims);
        std::vector<hsize_t> offset; //no offset
        histHPtr->writeToExistingHDF5Dataset(dset, offset);
        
        // Free
        delete histHPtr;
    }
}

template <typename T, unsigned int N>
void SpectralPolycrystalCUDA<T,N>::writeResultHDF5(std::string filename)
{
    H5::H5File outfile(filename.c_str(), H5F_ACC_TRUNC);
    
    // Stress history
    writeVectorToHDF5Array(outfile, "tHistory", this->tHistory);    
    std::vector<hsize_t> timeDims = {this->TCauchyHistory.size()};
    std::vector<hsize_t> tensorDims = {3,3};
    H5::DataSet TCauchyDset = createHDF5GridOfArrays<T>(outfile, "TCauchyHistory", timeDims, tensorDims);
    for (unsigned int i=0; i<this->TCauchyHistory.size(); i++) {
        std::vector<hsize_t> offset = {i};
        this->TCauchyHistory[i].writeToExistingHDF5Dataset(TCauchyDset, offset);
    }
    
    // Pole figure histograms
    std::vector<VecCUDA<T,3>> poles;
    poles.push_back(VecCUDA<T,3>{1,1,1});
    poles.push_back(VecCUDA<T,3>{1,1,0});
    poles.push_back(VecCUDA<T,3>{1,0,0});
    poles.push_back(VecCUDA<T,3>{0,0,1});
    poles.push_back(VecCUDA<T,3>{0,1,1});
    this->writePoleHistogramsHDF5(outfile, "poleHistograms", poles);
    
    // Scalar attributes
    addAttribute(outfile, "spectralPolycrystalSolveTime", solveTimer.getDuration());
    addAttribute(outfile, "nTimestepsTaken", this->getNTimestepsTaken());
    addAttribute(outfile, "nComponents", this->getNComponents());
    addAttribute(outfile, "nFourierTermsComputedHardware", this->getNTermsComputedHardware());
    addAttribute(outfile, "maxMemUsedGB", maxMemUsedGB);
 
    // Close
    outfile.close();
}

template <typename T, unsigned int N>
unsigned int SpectralPolycrystalCUDA<T,N>::getNTimestepsTaken() {
    return tHistory.size()-1;
}

template <typename T, unsigned int N>
unsigned int SpectralPolycrystalCUDA<T,N>::getNComponents() {
    if (useUnifiedDB) {
        return dbUnifiedH.getNDsets();
    }
    else {
        return dbH.getNDsets();
    }
}

template <typename T, unsigned int N>
unsigned long long int SpectralPolycrystalCUDA<T,N>::getNTermsComputedHardware() {
    if (useUnifiedDB) {
        unsigned long long int nTerms = 1;
        nTerms *= this->getNTimestepsTaken();
        nTerms *= nCrystals;  
        nTerms *= this->getNComponents();
        nTerms *= dbUnifiedH.getNTerms();
        return nTerms;
    }
    else {
        ///@todo implement
        return 0;
    }
}

// Explicit instantiations
template class SpectralCrystalListCUDA<float>;
template class SpectralCrystalListCUDA<double>;
template class CrystalPropertiesCUDA<float,12>;
template class CrystalPropertiesCUDA<double,12>;
template class SpectralPolycrystalCUDA<float,12>;
template class SpectralPolycrystalCUDA<double,12>;

#endif /* HPP_USE_CUDA */
}//END NAMESPACE HPP